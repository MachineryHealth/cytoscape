void writeOutput(graph * g){
	
	float zero = 0;
	float one = 1;
	FILE * oEdges =fopen("data\\4970_edge_6455.oedge","w");
	if(!oEdges) error("cannot open edge file");
	
	for(int i = 0; i < g->numVertices; i++)
		for(int j = g->AdjMatIndex[i]; j < g->AdjMatIndex[i+1]; j++)
			{
			int e2 = g->AdjMatVals[j];
			if(i < e2)
				fprintf(oEdges, "%d %d \r",i,e2);
			}
	fclose(oEdges);
	
	FILE * oVertices =fopen("data\\4970_edge_6455.ocoordinate","w");
	if(!oVertices) error("cannot open coordinate file");
	
	for(int i = 0; i < g->numVertices; i++)
		fprintf(oVertices, "%f %f %f\r",g->NodePos[i].x,g->NodePos[i].y,zero);
		
	fclose(oVertices);
	
	FILE * oGraph =fopen("data\\4970_edge_6455.graph","w");
	if(!oGraph) error("cannot open main.graph file");
	
	fprintf(oGraph, "./data/4970_edge_6455.ocoordinate\n");
	fprintf(oGraph, "./data/4970_edge_6455.oedge\n");
	fprintf(oGraph, "./data/4970_edge_6455.oweight\n");
	
	fclose(oGraph);
	
	FILE * oWeight =fopen("data\\4970_edge_6455.oweight","w");
	if(!oWeight) error("cannot open weight file");
	
	for(int i = 0; i < g->numVertices; i++)
		fprintf(oWeight, "%f\n",one);
		
	fclose(oWeight);
	
}

