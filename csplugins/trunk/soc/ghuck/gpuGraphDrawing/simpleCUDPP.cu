#include "hip/hip_runtime.h"
#include "licence.h"
#include "copyright.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cmath>
#include <ctime>

// includes, project
#include "cutil.h"
#include "GL/glut.h"
#include "cudpp.h"


// includes, kernels
#include <kernel.cu>

// Include other source files
#include "grap.cu"
#include "kdNode.cu"
#include "pkdNode.cu"
#include "common.h"
#include "readFile.cu"
#include "writeOutput.cu"


graph		g;
kdNodeInt *	rootInt;
kdNodeFloat *	rootFloat;
kdNodeInt *	treeIntD;  
kdNodeFloat *	treeFloatD; 
float2 *	NodePosD; 
float3 *	NodeTemp;
dim3		threads,blocks; 
int *		AdjMatIndexD;  
int *		AdjMatValsD;  
int *		edgeLenD;  
float2 *	DispD, * Disp; 
graph *		gArray[150] = {0};
int		numLevels;
int		coarseGraphSize;
int		interpolationIterations;
int		levelConvergence;
float3 *	a;
//CUDPPScanConfig config;                          Deprecated
CUDPPConfiguration config; // This struct replaces the old CUDPPScanConfig in the release 1.0a of CUDPP
unsigned int *	data_out;
unsigned int *	d_temp_addr_uint; 
float3 *	d_out;
unsigned int *	nD;
complexDevice * OuterD;
     

// This function calculates one step of the force-driven layout process, updating the nodes position
void advancePositions(graph * g)
{
  hipMemcpyToSymbol(HIP_SYMBOL(gd), g, sizeof(graph));

  // check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");	
  
  for(int i = 0; i < g->numVertices; i++){
    NodeTemp[i].x=g->NodePos[i].x;
    NodeTemp[i].y=g->NodePos[i].y;
    NodeTemp[i].z = i;
  }
  
  hipMemcpy(a, NodeTemp, g->numVertices*sizeof(float3), hipMemcpyHostToDevice);
  
  //config.maxNumElements = g->numVertices;        Deprecated
  //cudppInitializeScan(&config);                  Deprecated
      
  // Configure CUDPP Scan Plan
  CUDPPHandle planHandle;
  cudppPlan (&planHandle, config, g->numVertices, 1, 0); // rows = 1, rowPitch = 0
  
  int sizeInt = g->numVertices*sizeof(kdNodeInt);
  int sizeFloat = g->numVertices*sizeof(kdNodeFloat);
  
  // Check if the KDTREE has to be rebuilded
  if((g->currentIteration < 4) ||(g->currentIteration%20==0) ){

    // Decide whether the KDTREE is goint to be builded in the CPU or in the GPU
    if (g->numVertices < 50000){ //CPU
      kdNodeInit(rootInt,rootFloat,1,0,0,SCREEN_W,0,SCREEN_H);
      construct(NodeTemp, NodeTemp+g->numVertices-1, rootInt,rootFloat, 1,0,0,SCREEN_W,0,SCREEN_H,3);
    }
    else{ //GPU   
      kdNodeInitD(rootInt,rootFloat,1,0,0,SCREEN_W,0,SCREEN_H);
      constructD(a, a+g->numVertices-1, rootInt,rootFloat, 1,0,0,SCREEN_W,0,SCREEN_H,3,data_out,d_temp_addr_uint, d_out,planHandle,nD,OuterD );
    }
  }
  	
  hipMemcpy(NodePosD, g->NodePos, g->numVertices*sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(treeIntD, rootInt, sizeInt, hipMemcpyHostToDevice);
  hipMemcpy(treeFloatD, rootFloat, sizeFloat, hipMemcpyHostToDevice);
  hipBindTexture(0,texNodePosD, NodePosD,sizeof(float2)*g->numVertices);
  hipBindTexture(0,texInt, treeIntD,sizeInt);
  hipBindTexture(0,texFloat, treeFloatD,sizeFloat);
  
  hipMemcpy(AdjMatIndexD, g->AdjMatIndex, (g->numVertices+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(AdjMatValsD,  g->AdjMatVals,  (g->numEdges)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(edgeLenD,     g->edgeLen,     (g->numEdges)*sizeof(int), hipMemcpyHostToDevice);

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  hipBindTexture(0,texAdjMatValsD, AdjMatValsD,(g->numEdges)*sizeof(int));
  hipBindTexture(0,texEdgeLenD,edgeLenD,  (g->numEdges)*sizeof(int));

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
    
  // Execute the kernel, calculate forces
  calculateForces<<< blocks, threads >>>(g->numVertices, DispD,AdjMatIndexD);
  
  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");

  hipMemcpy(Disp, DispD, g->numVertices*sizeof(float2), hipMemcpyDeviceToHost);
	
  // Calculate new positions of nodes, based on the force calculations
  for(int i = 0; i < g->numVertices; i++)
    calcPositions(i,g->NodePos, Disp,g); 

  // Decrease the temperature of graph g
  cool(g); 

  // Destroy CUDPP Scan Plan
  cudppDestroyPlan(planHandle);
}


// This function coarses a graph, by obtaining a maximal independant subset of it
graph* coarsen(graph * g)
{
  graph *	rg = (graph*) malloc(sizeof(graph));
  bool *	used   = (bool*) calloc(g->numVertices,sizeof(bool));
  int *		newNodesNos = (int*) calloc(g->numVertices+1,sizeof(int));
  int		current = 0;
  int		left = g->numVertices;
  int		numParents = 0;
  rg->parent = (int *)  calloc(g->numVertices,sizeof(int));
  
  while(left>0){
    left--;
    newNodesNos[numParents] = current;
    rg->parent[current] = numParents;
    used[current] = 1;
    
    for(int x = g->AdjMatIndex[current]; x < g->AdjMatIndex[current+1]; x++){
      int j = g->AdjMatVals[x];
      if(!used[j])
	left --;
      used[j] = 1;
      rg->parent[j] = numParents;
    }
    numParents++;

    // If there is any node left, search for an unused one
    if(left>0)
      while((used[current]))
	current++;
  }
  
  free(used);
  
  initGraph(rg,numParents);
  int numEdges = 0;
  rg->NodePos = (float2 *) malloc((numParents)*sizeof(float2));
  rg->AdjMatIndex =  (int * )  calloc(numParents+1,sizeof(int));
  rg->AdjMatVals  =  (int * )  calloc(g->numEdges,sizeof(int));
  rg->edgeLen     =  (int * )  calloc(g->numEdges,sizeof(int));
  
  for(int i = 0; i < numParents; i++){
    rg->NodePos[i].x = rand()%SCREEN_W;
    rg->NodePos[i].y = rand()%SCREEN_H;
  }
  
  for ( int i = 0; i < numParents; i++){
    int * usedChild = (int *) calloc(numParents,sizeof(int));
    int node = newNodesNos[i];
    for(int x = g->AdjMatIndex[node]; x < g->AdjMatIndex[node+1]; x++){
      int j = g->AdjMatVals[x];
      if (rg->parent[j] != i)
	usedChild[rg->parent[j]] = 1;
      else{
	for(int y = g->AdjMatIndex[j]; y < g->AdjMatIndex[j+1]; y++){
	  int neighbor = g->AdjMatVals[y];
	  usedChild[rg->parent[neighbor]] = 1;
	}
      }
    }
    
    for ( int k = 0; k < numParents; k++){
      if (usedChild[k]){
	rg->AdjMatVals[numEdges] = k;
	rg->edgeLen[numEdges] = EDGE_LEN;
	numEdges++;
      }
    }
      
    rg->AdjMatIndex[i+1] = numEdges;
    free(usedChild);
  }  
  
  rg->numEdges = numEdges;
  return rg;
}


// This function just applies a one step advance to a graph position
void exactLayoutOnce(graph * g){
  advancePositions(g);
}

// This funcion initializes a graph position, using the position of nodes in the coarsed graph (if it exists) as a guide
void nextLevelInitialization(graph g, graph * coarseGraph){
  
  // Nodes that exists in coarseGraph remain in the same position
  for(int i = 0; i < g.numVertices; i++){
    g.NodePos[i].x = coarseGraph->NodePos[coarseGraph->parent[i]].x ;
    g.NodePos[i].y = coarseGraph->NodePos[coarseGraph->parent[i]].y ;
  }
  
  //
  for(int j = 0; j <interpolationIterations; j++){
    for(int i = 0; i < g.numVertices; i++){
      int degree = g.AdjMatIndex[i+1] - g.AdjMatIndex[i];
      float2 pi; pi.x=0;pi.y=0;
      for(int k = g.AdjMatIndex[i]; k < g.AdjMatIndex[i+1]; k++){	
	int j = g.AdjMatVals[k];
	pi.x+=g.NodePos[j].x;
	pi.y+=g.NodePos[j].y;
      }
      if(degree){
	g.NodePos[i].x = 0.5 * ( g.NodePos[i].x+ (1.0/degree)*pi.x);
	g.NodePos[i].y = 0.5 * ( g.NodePos[i].y+ (1.0/degree)*pi.y);
      }
    }
  }
  
  
  free(coarseGraph->NodePos);
  free(coarseGraph->parent);
  free(coarseGraph->AdjMatIndex);
  free(coarseGraph->AdjMatVals);
  free(coarseGraph->edgeLen);
  free(coarseGraph);
}

// This function creates the MIS (Maximal Independent Set) Filtration of a graph
void createCoarseGraphs(graph * g,int level)
{
  gArray[level] = g;
  if(g->numVertices <= coarseGraphSize)
    return;
  
  graph * coarseGraph = coarsen(g);
  
  if(g->numVertices < 1.07 * coarseGraph->numVertices )
    return;
  
  if(g->numVertices - coarseGraph->numVertices > 0 )
    createCoarseGraphs(coarseGraph,level+1);
}

// Show results in screen 
void display(void)
{	
  glLoadIdentity();
  glClearColor(1.0f, 1.0f, 1.0f, 1.0f);	
  int l = 0;
  
  glClear(GL_COLOR_BUFFER_BIT);
  glLoadIdentity();
  glBegin(GL_LINES);
  glColor3f(0.2,0.2,0.2);
  for(int i = 0; i < gArray[l]->numVertices; i++)
    for(int j = gArray[l]->AdjMatIndex[i]; j < gArray[l]->AdjMatIndex[i+1]; j++){
      int k = gArray[l]->AdjMatVals[j];
      glVertex3f(gArray[l]->NodePos[i].x,gArray[l]->NodePos[i].y,00);
      glVertex3f(gArray[l]->NodePos[k].x,gArray[l]->NodePos[k].y,00);
    }
  glEnd();
  glColor3f(1,0,0);
  glPointSize(1.1);
  glBegin(GL_POINTS);
  for(int i = 0; i < gArray[l]->numVertices; i++)
    glVertex3f(gArray[l]->NodePos[i].x,gArray[l]->NodePos[i].y,00);
  glEnd();
  
  glFlush();  /* OpenGL is pipelined, and sometimes waits for a full buffer to execute */
  glutSwapBuffers();
}

// Reshape screen
void reshape(int w,int h)
{
  glViewport(0,0,w,h);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
  // Initialize device, using macro defined in "cutil.h"
  CUT_DEVICE_INIT();
  
  FILE* from;
  graph g;

  // Check number of arguments
  if (argc < 2) error("Wrong no of args");

  // Ask for parameters
  printf("Enter the size of the coarsest graph (Default 50):"); scanf("%d",&coarseGraphSize);
  printf("Enter the number of interpolation iterations (Default 50):"); scanf("%d", &interpolationIterations);
  printf("Enter the level of convergence (Default 2):"); scanf("%d",&levelConvergence);
  printf("Enter the ideal edge length (Default 5):"); scanf("%d",&EDGE_LEN);
  printf("Enter the initial no of force iterations(Default 300):"); scanf("%d",&initialNoIterations);
 
  // Open file 
  from=fopen(argv[1],"r");
  if(!from) error("cannot open 1st file");
  
  //Read graph grom file (argv[1])
  int len = strlen(argv[1]);
  if((argv[1][len-1]=='l') && (argv[1][len-2]=='m') && (argv[1][len-3]=='g') )
    readGml(&g, from);
  else
    readChaco(&g, from);
  
  /*    Initializations    */

  // Number of Nodes
  int  numNodes = g.numVertices;

  // Amount of memory to be used by integers
  int sizeInt = numNodes*sizeof(kdNodeInt);

  // Amount of memory to be used by integers
  int sizeFloat = numNodes*sizeof(kdNodeFloat);
  
  rootInt   = (kdNodeInt *) calloc(numNodes,sizeof(kdNodeInt));
  rootFloat = (kdNodeFloat *) calloc(numNodes,sizeof(kdNodeFloat));
  hipMalloc((void**)&treeIntD,sizeInt);
  hipMalloc((void**)&treeFloatD,sizeFloat);
  hipMalloc((void**)&NodePosD, numNodes*sizeof(float2));
  
  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  // check if kernel execution generated and error
  NodeTemp = (float3 *)malloc(numNodes*sizeof(float3));
  hipMalloc((void**)&a, numNodes*sizeof(float3));
  
  Disp = (float2 *) malloc((numNodes)*sizeof(float2));
  hipMalloc((void**)&DispD, numNodes*sizeof(float2));
  
  hipMalloc((void**)&AdjMatIndexD, (g.numVertices+1)*sizeof(int));
  hipMalloc((void**)&AdjMatValsD, (g.numEdges)*sizeof(int));
  hipMalloc((void**)&edgeLenD, (g.numEdges)*sizeof(int));
  
  // Initialize parameters for config (see CUDPP in cudpp.h)

  config.algorithm = CUDPP_SCAN;
  config.op        = CUDPP_ADD;
  config.datatype  = CUDPP_INT;
  config.options   = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE; 

  //config.direction      = CUDPP_SCAN_FORWARD;                  Deprecated
  //config.exclusivity    = CUDPP_SCAN_EXCLUSIVE;                Deprecated
  //config.op	          = CUDPP_ADD;                           Deprecated
  //config.datatype       = CUDPP_INT;                           Deprecated
  //config.maxNumRows	  = 1;                                   Deprecated
  //config.rowPitch       = 0;                                   Deprecated


  
  hipMalloc((void**)&data_out,sizeof(unsigned int)* g.numVertices);
  hipMalloc((void**)&d_temp_addr_uint,sizeof(unsigned int)* g.numVertices);
  hipMalloc((void**)&d_out,sizeof(float3)* g.numVertices);
  hipMalloc((void**)&nD,sizeof(unsigned int));
  /* End Initializations */
  
  
  printf("Coarsening graph...\n");
  
  clock_t start, end_coarsen,end_layout;
  double elapsed_layout,elapsed_coarsen;
  start = clock();
  
  
  gArray[0] = &g;
  createCoarseGraphs(&g,0);
  numLevels=0;
  while(gArray[numLevels]!=NULL)
    numLevels++;
  gArray[numLevels-1]->level = 0;
  
  end_coarsen = clock();
  elapsed_coarsen = ((double) (end_coarsen - start)) / CLOCKS_PER_SEC;
  start = clock();
  printf("Computing layout...\n");
  
  for(int i = 0; i < numLevels; i++){
    
    // setup execution parameters
    
    unsigned m_chunks = gArray[numLevels-i-1]->numVertices / maxThreadsThisBlock;
    unsigned m_leftovers = gArray[numLevels-i-1] ->numVertices % maxThreadsThisBlock;
    
    if ((m_chunks == 0) && (m_leftovers > 0)){
      // can't even fill a block
      blocks = dim3(1, 1, 1); 
      threads = dim3((m_leftovers), 1, 1);
    } 
    else {
      // normal case
      if (m_leftovers > 0){
	// not aligned, add an additional block for leftovers
	blocks = dim3(m_chunks + 1, 1, 1);
      }
      else{
	// aligned on block boundary
	blocks = dim3(m_chunks, 1, 1);
      }
      threads = dim3(maxThreadsThisBlock , 1, 1);
    }
    
    if(i < numLevels-levelConvergence)
      while(!incrementsAreDone(gArray[numLevels-i-1]))
	exactLayoutOnce(gArray[numLevels-i-1]);
    if(numLevels-i-2 >= 0)
      nextLevelInitialization(*gArray[numLevels-i-2], gArray[numLevels-i-1]);
  }
  
  end_layout = clock();
  elapsed_layout = ((double) (end_layout - start)) / CLOCKS_PER_SEC;
  
  printf("Time for coarsening graph: %f\n",elapsed_coarsen);
  printf("Time for calculating layout: %f\n",elapsed_layout);
  
  hipFree(AdjMatIndexD);
  hipFree(edgeLenD);
  hipFree(AdjMatValsD);
  hipFree(NodePosD);
  hipFree(DispD);
  hipFree(treeIntD);
  hipFree(treeFloatD);
  hipFree(data_out);
  hipFree(d_temp_addr_uint);
  hipFree(d_out);
  hipFree(nD);
  free(NodeTemp);
  free(rootInt);
  free(rootFloat);
  free(Disp);
  
  writeOutput(&g);
  
  
  glutInit(&argc, argv);		/* setup GLUT */
  glutInitDisplayMode(GLUT_RGB); 
  glutInitWindowSize(SCREEN_W,SCREEN_H);
  glutInitWindowPosition(100,100);
  glutCreateWindow(argv[0]);	/* open a window */
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluOrtho2D(0,SCREEN_W,0,SCREEN_H);
  glMatrixMode(GL_MODELVIEW);
  glutReshapeFunc(reshape);
  glutDisplayFunc(display);		/* tell GLUT how to fill window */
  glutMainLoop();				/* let glut manage i/o processing */
  
  return EXIT_SUCCESS;
}
