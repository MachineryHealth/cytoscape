#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"


void error(const char * p, const char * p2="")
{
  printf("%s %s\n",p,p2);
  exit(1);
}
			    

// This function reads a graph from a file (from) stored in a quite strange GML format
//TODO: FIX free() call in readGml() function. It produces a segmentation fault
// TODO: Write a function which reads from standard GML format
			  


/******************************* FORMAT SAMPLE ***********************
graph [
 Creator "makegml" directed 0 label ""
  node [ id 1 ]
  node [ id 2 ]
  node [ id 3 ]
  node [ id 4 ]
  node [ id 5 ]
  node [ id 6 ]
edge [ source 1 target 3 ]
edge [ source 1 target 4 ]
edge [ source 2 target 2 ]
edge [ source 2 target 3 ]
edge [ source 3 target 1 ]
edge [ source 3 target 2 ]
edge [ source 4 target 5 ]
]
**************************************************************************/


void readGml(graph * g, FILE * from)
{
  int numNodes;
  char string[MAX_REC_LEN]; // used to store data readed from the file temporaly

  printf("Reading nodes!");

  // Skip first 2 lines
  fgets(string, MAX_REC_LEN,from ); // graph [
  fgets(string, MAX_REC_LEN,from ); // Creator "makegml" directed 0 label ""


  /*     READ NODES    */

  // Get the offset (starting position)
  fgets(string, MAX_REC_LEN,from ); 

  int i=0;
  while(string[i]!='[')
    i++;
  int startPos = i; 

  // Get number of nodes
  numNodes = atoi(string+startPos+4);       // get initial node number
  while( fgets(string, MAX_REC_LEN,from) ){ // get a new line
    int n = atoi(string+startPos+4);        // read node number
    if(n!=numNodes+1)                       // check whether the new node number is numNodes+1 (if not, we're done here)
      break;
    else  
      numNodes = n;
    //printf("!Node:%ld!\n",ftell (from));		
  }
  printf ("\tnumNodes = %d\n", numNodes);

  /*     FINISH READING NODES      */


  // Get the position in the file in which the edge information starts (secFrom)
  long int secFrom  = ftell (from);         
  secFrom -= (long int)(strlen(string)+1);

  // Allocate memory for NodePos
  g->NodePos = (float2*) malloc((numNodes)*sizeof(float2));

  // Allocate memory for AdjMatIndex 
  g->AdjMatIndex =  (int*) calloc((numNodes+1),sizeof(int));


  /*     READ EDGES      */

  g->AdjMatIndex[0]=0;                       // Adjacency list of first node starts in position 0
  int numEdges = 0;                          // Initialize numEdges

  printf("Reading edges!"); 

  // Get the offset (starting position) for edge numbers
  i = 0;
  while(string[i]!='[')
    i++;
  startPos = i;

  // Get the edge source node (e1)
  int e1 = atoi(string+startPos+9 );

  // Go to the position of edge target node
  i=startPos+9;
  while(string[i]!= 't')
    i++;

  // Get the edge target node (e2)
  int e2 = atoi(string+i+ 6);

  // Increase number of edges, Adj Matrix indexes of e1 and e2
  (g->AdjMatIndex[e1])++;
  (g->AdjMatIndex[e2])++;
  numEdges++;
  
  // Process the rest of the edges
  while(fgets(string, MAX_REC_LEN,from )){
    
    // Check if the file is finishing
    if((string[0]==']') || (string[1]==']'))
      break;

    // Increase number of edges
    numEdges++;

    // Get the edge source node (e1)
    e1 = atoi(string+startPos+9 );

    // Go to the position of edge target node
    i=0;
    while(string[i]!= 't')
      i++;

    // Get the edge target node (e2)
    int e2 = atoi(string+i+ 6);
    (g->AdjMatIndex[e1])++;
    (g->AdjMatIndex[e2])++;
  }
  
  // Update AdjMatIndex so that each position points to the appropiate element in AdjMatVals
  for(int i = 0; i < numNodes; i++)
    g->AdjMatIndex[i+1] += g->AdjMatIndex[i];
  
  printf("\tNumber of Edges = %d\n",numEdges);

  // Allocate memory for AdjMatVals, edgeLen
  g->AdjMatVals  = (int*) malloc(2*numEdges*sizeof(int));
  g->edgeLen     = (int*) malloc(2*numEdges*sizeof(int));

  // Allocate memory for temp, an auxiliary array, initialize it whith zeros
  int *temp      = (int*) calloc(numNodes,sizeof(int));

  // Initialize Graph
  initGraph(g,numNodes); 

  // Save numEdges
  g->numEdges = 2*numEdges;

  // Go to secFrom position in file "from" (where the edge information starts)
  fseek ( from, secFrom, SEEK_SET );
  while( fgets(string, MAX_REC_LEN,from )){
    
    // Check if the file is finishing
    if( (string[0]==']') || (string[1]==']') )
      break;

    // Get the edge source node (e1)
    e1 = atoi(string+startPos+9 );

    // Go to the position of edge target node
    i=0;
    while(string[i]!= 't')
      i++;

    // Get the edge target node (e2)
    int e2 = atoi(string+i+ 6);

    // Add e1 to adjacency list of e2 and vice versa.
    g->AdjMatVals[g->AdjMatIndex[e1-1]+temp[e1-1]] = e2-1;
    g->AdjMatVals[g->AdjMatIndex[e2-1]+temp[e2-1]] = e1-1;

    // Save edge lenght for this edge
    g->edgeLen[g->AdjMatIndex[e1-1]+temp[e1-1]] = EDGE_LEN;
    g->edgeLen[g->AdjMatIndex[e2-1]+temp[e2-1]] = EDGE_LEN;

    // Increase the number of neighbors already processed of e1 and e2 
    (temp[e1 - 1])++;
    (temp[e2 - 1])++;

  }
 
  // FIX IT! The following free() call gives segmentation fault
  //  free ((void*) temp);
}





/******************************* FORMAT SAMPLE ***********************
7 10
2 3
1 3 7
1 2 6 7
6
6 7
3 4 5 7
2 3 5 6
**********************************************************************/
//In more detail, there are 7 nodes and 10 edges in the graph; node 1 is adjacent to 2,3; node 2 is adjacent to 1,3,7; etc


void readChaco(graph * g, FILE * from)
{
  int numNodes,numEdges;
  char string[MAX_REC_LEN];        // Temporary string in which each line of the file will be temporary stored
  int index = 0;
  int nEdges = 0;
  
  if(!fscanf(from,"%d",&numNodes))
    error("Cannot read 1st file");
  if(!fscanf(from,"%d",&numEdges))
    error("Cannot read 1st file");

  printf ("Number of nodes: %d\n", numNodes);
  printf ("Number of edges: %d\n", numEdges);
  printf ("Reading nodes!!\n");		
  printf ("Reading edges!!\n");

  // Initialize Graph
  initGraph(g,numNodes); 

  // Save numEdges
  g->numEdges = 2*numEdges;

  // Allocate memory for NodePos, AdjMatIndex, AdjMatVals, edgeLen
  g->NodePos     = (float2*) malloc((numNodes)*sizeof(float2));
  g->AdjMatIndex =    (int*) malloc((numNodes+1)*sizeof(int));
  g->AdjMatVals  =    (int*) malloc(2*numEdges*sizeof(int));
  g->edgeLen     =    (int*) malloc(2*numEdges*sizeof(int));
	
  // First node's adjacency list starts at position 0
  g->AdjMatIndex[0]=0;
  while(fgets(string, MAX_REC_LEN,from )){
    
    if((string[0]==10) || (string[0]==8) ) 
      continue;
    
    g->NodePos[index].x= (int)rand()%g->screen_width;
    g->NodePos[index].y= (int)rand()%g->screen_hieght;
    
    if(g->NodePos[index].x < 0){
      exit(0);
    }
    
    char * first = string;
    int sl=strlen(string);
    
    for(int i=0;i<sl;i++){
    
      if(string[i]==10){
	string[i]='\0';
	int n = atoi(first);
	first = &string[i]; 
	g->AdjMatVals[nEdges] = n - 1;
	g->edgeLen[nEdges] = EDGE_LEN;
	nEdges++;
	break;
      }
      
      if(i==0 && string[0]==' '){
	first=string+1;
	continue;
      }
      if(string[i]!=' ') continue;
      
      while(string[i]==' ') 
	i++;
      
      string[i-1]='\0';
      
      if(strlen(first)){
	int n = atoi(first);
	first = &string[i]; 
	g->AdjMatVals[nEdges] = n - 1;
	g->edgeLen[nEdges] = EDGE_LEN;
	nEdges++;
      }
    }
    
    g->AdjMatIndex[index+1] = nEdges;
    index++;
    
  }	
}

