#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See licence.h for more information.
**************************************************************************************/
#include "license.h"


void error(const char * p, const char * p2="")
{
	printf("%s %s\n",p,p2);
	exit(1);
}

void readGml(graph * g, FILE * from){
	
	printf("Reading nodes!!\n");
	int numNodes;
	char string[MAX_REC_LEN];
	fgets(string, MAX_REC_LEN,from ); // graph [
	fgets(string, MAX_REC_LEN,from ); //  Creator "makegml" directed 0 label ""
	fgets(string, MAX_REC_LEN,from ); 
	int i=0;
	while(string[i]!='[')
		i++;
	int startPos = i; 
	numNodes = atoi(string+startPos+4);
	while(fgets(string, MAX_REC_LEN,from )){
		int n = atoi(string+startPos+4);
		if(n!=numNodes+1)
			break;
		else  
			numNodes = n;
		//printf("!Node:%ld!\n",ftell (from));		
	}
	long int secFrom  = ftell (from);
	secFrom -= (long int)(strlen(string)+1);
	
	g->NodePos = (float2 *) malloc((numNodes)*sizeof(float2));
	g->AdjMatIndex =  (int * )  calloc((numNodes+1),sizeof(int));
	g->AdjMatIndex[0]=0;
	int numEdges = 0;
	printf("Reading edges!!\n"); 
	i = 0;
	while(string[i]!='[')
		i++;
	startPos = i;
	int e1 = atoi(string+startPos+9 );
	i=startPos+9;
	while(string[i]!= 't')
		i++;
	int e2 = atoi(string+i+ 6);
	g->AdjMatIndex[e1]++;
	g->AdjMatIndex[e2]++;
	numEdges++;
	
	while(fgets(string, MAX_REC_LEN,from )){
		if((string[0]==']') || (string[1]==']'))
			break;
		numEdges++;
		e1 = atoi(string+startPos+9 );
		i=0;
		while(string[i]!= 't')
		i++;
		int e2 = atoi(string+i+ 6);
		g->AdjMatIndex[e1]++;
		g->AdjMatIndex[e2]++;
	}
	for(int i = 0; i < numNodes; i++)
		g->AdjMatIndex[i+1] += g->AdjMatIndex[i];
			
	printf("No of Edges: %d\n",numEdges);
	g->AdjMatVals  =  (int * )  malloc(2*numEdges*sizeof(int));
	g->edgeLen     =  (int * )  malloc(2*numEdges*sizeof(int));
	int * temp =  (int * )  calloc((numNodes),sizeof(int));
	initGraph(g,numNodes); g->numEdges = 2*numEdges;
	
	fseek ( from, secFrom, SEEK_SET );
	while(fgets(string, MAX_REC_LEN,from )){
		
		if((string[0]==']') || (string[1]==']'))
			break;
		e1 = atoi(string+startPos+9 );
		i=0;
		while(string[i]!= 't')
			i++;
		int e2 = atoi(string+i+ 6);
		g->AdjMatVals[g->AdjMatIndex[e1-1]+temp[e1-1]] = e2-1;
		g->AdjMatVals[g->AdjMatIndex[e2-1]+temp[e2-1]] = e1-1;
		g->edgeLen[g->AdjMatIndex[e1-1]+temp[e1-1]] = EDGE_LEN;
		g->edgeLen[g->AdjMatIndex[e2-1]+temp[e2-1]] = EDGE_LEN;
		temp[e1-1]++;
		temp[e2-1]++;
	}
	free(temp);
	
}


void readChaco(graph * g, FILE * from){
	int numNodes,numEdges;
	printf("Reading nodes!!\n");
	if(!fscanf(from,"%d",&numNodes))
		error("Cannot read 1st file");
	if(!fscanf(from,"%d",&numEdges))
		error("Cannot read 1st file");
		
	printf("Reading edges!!\n");
	char string[MAX_REC_LEN];
	int index = 0;
	initGraph(g,numNodes); g->numEdges = 2*numEdges;
	g->NodePos = (float2 *) malloc((numNodes)*sizeof(float2));
	g->AdjMatIndex =  (int * )  malloc((numNodes+1)*sizeof(int));
	g->AdjMatVals  =  (int * )  malloc(2*numEdges*sizeof(int));
	g->edgeLen     =  (int * )  malloc(2*numEdges*sizeof(int));
	
	int nEdges = 0;
	g->AdjMatIndex[0]=0;
	while(fgets(string, MAX_REC_LEN,from ))
	{
		if((string[0]==10) || (string[0]==8) ) continue;
		g->NodePos[index].x= (int)rand()%g->screen_width;
		g->NodePos[index].y= (int)rand()%g->screen_hieght;
		if(g->NodePos[index].x < 0)
		{
		exit(0);
		}
		char * first = string;
		int sl=strlen(string);
		for(int i=0;i<sl;i++)
		{
			if(string[i]==10)
			{
				string[i]='\0';
				int n = atoi(first);
				first = &string[i]; 
				g->AdjMatVals[nEdges] = n - 1;
				g->edgeLen[nEdges] = EDGE_LEN;
				nEdges++;
				break;
			}
			if(i==0 && string[0]==' '){
				first=string+1;
				continue;
			}
			if(string[i]!=' ') continue;
			while(string[i]==' ') 
					i++;
					string[i-1]='\0';
			if(strlen(first)){
			int n = atoi(first);
			first = &string[i]; 
			g->AdjMatVals[nEdges] = n - 1;
			g->edgeLen[nEdges] = EDGE_LEN;
			nEdges++;
			}
		}
		g->AdjMatIndex[index+1] = nEdges;
		index++;
	}	
}

