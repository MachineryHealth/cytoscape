#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
// GPLv3 License
#include "license.h"

// Header with JNI's function declaration
#include "../GpuLayout.h" 

// GpuGraphDrawing interface
#include "interface.cu"




////////////////////////////////////////////////////////////////////////////////
// 
////////////////////////////////////////////////////////////////////////////////

JNIEXPORT jobjectArray JNICALL Java_GpuLayout_ComputeGpuLayout (JNIEnv*    env, 
								jobject    thisJ, 
								jintArray  AdjMatIndexJ, 
								jintArray  AdjMatValsJ, 
								jint       coarseGraphSizeJ, 
								jint       interpolationIterationsJ, 
								jint       levelConvergenceJ, 
								jint       EDGE_LENJ, 
								jint       initialNoIterationsJ, 
								jdouble    hSizeJ, 
								jdouble    vSizeJ
								)
{

  // Scope which will be used for this computation
  globalScope *scope;

  // Create scope
  scope = globalScopeCreate();

  // Set parameters
  scope->coarseGraphSize          = coarseGraphSizeJ;
  scope->interpolationIterations  = interpolationIterationsJ;
  scope->levelConvergence         = evelConvergenceJ;
  scope->EDGE_LEN                 = EDGE_LENJ;
  scope->initialNoIterations      = initialNoIterationsJ;

  ////////////
  // Set graph
  ////////////
  int numNodes,numEdges;
  int index = 0;
  int nEdges = 0;

  // Get numNodes, numEdges
  numNodes = env->getArrayLength(AdjMatIndexJ) - 1; //AdjMatIndexJ has an extra index for marking the end of AdjMatValsJ
  numEdges = env->getArrayLength(AdjMatValsJ);

  // Initialize Graph
  initGraph(&(scope->g), numNodes); 

  // Save numEdges
  scope->g.numEdges = numEdges;

  // Allocate memory for NodePos, AdjMatIndex, AdjMatVals, edgeLen
  scope->g.NodePos     = (float2*) malloc (numNodes       * sizeof(float2) );
  scope->g.AdjMatIndex =    (int*) malloc ((numNodes + 1) * sizeof(int)    );
  scope->g.AdjMatVals  =    (int*) malloc (numEdges       * sizeof(int)    );
  scope->g.edgeLen     =    (int*) malloc (numEdges       * sizeof(int)    );

  // Get temporary copies of AdjMatIndex, AdjMatVals
  int *temp_AdjMatIndex = env->GetIntArrayElements(AdjMatIndexJ, NULL);
  int *temp_AdjMatVals  = env->GetIntArrayElements(AdjMatValsJ , NULL);
	
  // Copy temporary copies
  memcpy (scope->g.AdjMatIndex, temp_AdjMatIndex, (numNodes + 1) * sizeof(int));
  memcpy (scope->g.AdjMatVals,  temp_AdjMatVals,  (numEdges)     * sizeof(int));

  // Initialize node positions 
  for (int i = 0; i < numNodes; i++){
      scope->g.NodePos[i].x = (int)rand() % scope->g.screen_width;
      scope->g.NodePos[i].y = (int)rand() % scope->g.screen_hieght;
    }
	 
  // Initialize edge lengths
  for (int i = 0; i < scope->g.AdjMatIndex[numNodes]; i++){
      scope->g.edgeLen[i] = scope->EDGE_LEN;
    }			
	
			
  // Free graph in JVM
  env->ReleaseIntArrayElements

		
  // Calculate layout
  calculateLayout (scope);
	      
  // Show results in display
  showGraph (scope, 0, NULL);
    
  // Wait
  Sleep(3);

  // Save nodes positions 


  return NULL;
}
